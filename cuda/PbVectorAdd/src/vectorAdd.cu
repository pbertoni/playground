#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>
#include <iostream>
#include <chrono>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
d_vector_add(const float * A, const float * B, float * C, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements)
	{
		C[i] = A[i] + B[i];
	}
}


void print_last_error(const std::string & MSG, hipError_t & err)
{
	std::cerr << "Failed to " << MSG << ", error code: " << hipGetErrorString(err) << std::endl;
}


/**
 * Host main routine
 */
int main(void)
{
	// Problem parameters
	const int numElements = 500000 * 128;
	const size_t threadsPerBlock = 32;
	const size_t blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	// Print the vector length to be used, and compute its size
	const size_t size = numElements * sizeof(float);
	std::cout << "Vector addition of " << numElements << " elements" << std::endl;

	// Allocate the host input vector A, B, C
	float	* h_A = (float *) malloc(size),
			* h_B = (float *) malloc(size),
			* h_C = (float *) malloc(size);

	// Verify that allocations succeeded
	if (h_A == nullptr or h_B == nullptr or h_C == nullptr)
	{
		std::cerr << "Failed to allocate host vectors!" << std::endl;
		exit(EXIT_FAILURE);
	}

	// Initialize the host input vectors
	for (int i = 0; i < numElements; ++i)
	{
		h_A[i] = rand() / (float)RAND_MAX;
		h_B[i] = rand() / (float)RAND_MAX;
	}

	// Allocate the device input vectors A, B, C
	float * d_A = nullptr, * d_B = nullptr, * d_C = nullptr;

	err = hipMalloc((void **)&d_A, size);
	if (err != hipSuccess)
	{
		print_last_error("allocate device vector A", err);
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_B, size);
	if (err != hipSuccess)
	{
		print_last_error("allocate device vector B", err);
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_C, size);
	if (err != hipSuccess)
	{
		print_last_error("allocate device vector C", err);
		exit(EXIT_FAILURE);
	}


	const auto T0 = std::chrono::high_resolution_clock::now();
	// Input: copy the host vectors A and B from host memory to the device vectors in device memory
	std::cout << "Copy input data from the host memory to the CUDA device" << std::endl;

	err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		print_last_error("copy vector A from host to device", err);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		print_last_error("copy vector B from host to device", err);
		exit(EXIT_FAILURE);
	}


	// Launch the Vector Add CUDA Kernel
	std::cout << "CUDA kernel launch with " << blocksPerGrid << " blocks of "
		<< threadsPerBlock << " threads each" << std::endl;

	d_vector_add<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

	err = hipGetLastError();
	if (err != hipSuccess)
	{
		print_last_error("launch vectorAdd kernel", err);
		exit(EXIT_FAILURE);
	}


	// Output: copy the device vector in device memory to the host vector in host memory
	std::cout << "Copy output data from the CUDA device to the host memory" << std::endl;

	err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		print_last_error("copy vector C from device to host", err);
		exit(EXIT_FAILURE);
	}
	const auto T1 = std::chrono::high_resolution_clock::now();

	// Verify that the result vector is correct
	for (int i = 0; i < numElements; ++i)
	{
		if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
		{
			std::cerr << "Result verification failed at element " << i << std::endl;
			exit(EXIT_FAILURE);
		}
	}
	const auto T2 = std::chrono::high_resolution_clock::now();

	const auto T01 = std::chrono::duration_cast<std::chrono::microseconds>(T1 - T0).count(),
				T12 = std::chrono::duration_cast<std::chrono::microseconds>(T2 - T1).count();

	std::cout << "Test PASSED: " << T01 << " vs " << T12 << " microseconds" << std::endl;


	// Free device global memory
	err = hipFree(d_A);
	if (err != hipSuccess)
	{
		print_last_error("free device vector A", err);
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_B);
	if (err != hipSuccess)
	{
		print_last_error("free device vector B", err);
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_C);
	if (err != hipSuccess)
	{
		print_last_error("free device vector C", err);
		exit(EXIT_FAILURE);
	}

	// Free host memory
	free(h_A);
	free(h_B);
	free(h_C);

	std::cout << "program end with 0";
	return 0;
}

