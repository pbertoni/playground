/*
 ============================================================================
 Name        : Prints.cu
 Author      : Patrizio Bertoni
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>


static void CheckCudaErrorAux(const char *, unsigned, const char *, hipError_t);

#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)


/**
 * CUDA kernel that computes reciprocal values for a given vector
 */
__global__ void reciprocalKernel(float * data, unsigned vectorSize)
{
	const unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < vectorSize)
	{
		data[idx] = 1.0/data[idx];
	}
}


/**
 * CUDA kernel for prints
 */
__global__ void hw()
{
        printf("Hw %d\n", blockIdx.x);
}


/**
 * Host function that copies the data and launches the work on GPU
 */
float * gpuReciprocal(float * h_pfData, unsigned size)
{
	const size_t SZ = sizeof(float) * size;

	float * rc = new float[size];
	float * d_pfData;

	CUDA_CHECK_RETURN(hipMalloc((void **) &d_pfData, SZ));
	CUDA_CHECK_RETURN(hipMemcpy(d_pfData, h_pfData, SZ, hipMemcpyHostToDevice));

	const int BLOCK_SIZE = 256;
	const int BLOC_COUNT = (size+BLOCK_SIZE-1)/BLOCK_SIZE;

	reciprocalKernel<<<BLOC_COUNT, BLOCK_SIZE>>> (d_pfData, size);

	CUDA_CHECK_RETURN(hipMemcpy(rc, d_pfData, SZ, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipFree(d_pfData));

	return rc;
}


/**
 * Host function for reference
 */
float * cpuReciprocal(float * data, unsigned size)
{
	float * rc = new float[size];

	for (unsigned cnt = 0; cnt < size; ++cnt)
	{
		rc[cnt] = 1.0/data[cnt];
	}

	return rc;
}


void initialize(float *data, unsigned size)
{
	for (unsigned i = 0; i < size; ++i)
	{
		data[i] = .5 * (i+1);
	}
}


int main(void)
{
	const int WORK_SIZE = 65530;
	float * pfData = new float[WORK_SIZE];

	initialize(pfData, WORK_SIZE);

	const float * pfReciprocalCpu = cpuReciprocal(pfData, WORK_SIZE);
	const float * pfReciprocalGpu = gpuReciprocal(pfData, WORK_SIZE);

	const float fSumCpu = std::accumulate(pfReciprocalCpu, pfReciprocalCpu + WORK_SIZE, 0.0);
	const float fSumGpu = std::accumulate(pfReciprocalGpu, pfReciprocalGpu + WORK_SIZE, 0.0);

	// Verify the results
	std::cout << "fSumGpu = " << fSumGpu << " fSumCpu = " << fSumCpu <<std::endl;

	// Free
	delete[] pfData;
	delete[] pfReciprocalCpu;
	delete[] pfReciprocalGpu;

	// Prints
    hw<<<16, 1>>>();

    hipDeviceSynchronize();

	return 0;
}


/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char * file, unsigned line, const char * statement, hipError_t err)
{
	if (err == hipSuccess)
	{
		return;
	}
	else
	{
		std::cerr << statement <<" returned " << hipGetErrorString(err) << "("<< err
				<< ") at " << file << ":" <<line << std::endl;
		exit (1);
	}
}

