#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include "../include/Performances.hpp"

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>

#include <iostream>
#include <vector>


/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__
void
d_vector_add(const float * A, const float * B, float * C, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements)
	{
		C[i] = A[i] + B[i];
	}
}



class VectorTester
{
	const int m_numElements;
	const size_t SZ_VEC_BYTES;

	hipError_t m_err;

	std::vector<float> h_A, h_B, h_C;

	float * d_A, * d_B, * d_C;

	std::string _describe(const std::string & prefix);

public:

	VectorTester(int numElements);

	/**
	 * Free device global memory
	 */
	~VectorTester();

	/**
	 * Initialize the host input vectors
	 *
	 * TODO overload with outer values
	 */
	void randomize();

	/**
	 * Launch the Vector Add CUDA Kernel
	 */
	void launch();

	/**
	 *  Copy the device result vector in device memory to the host result vector in host memory.
	 */
	void check();

	/**
	 * Copy the host input vectors A and B in host memory to the device input vectors in device memory
	 */
	void load_values();

	/**
	 * Compute vectorAdd in-thread, on-CPU.
	 */
	void cpu_benchmark();
};


VectorTester::VectorTester(int numElements) :
	m_numElements(numElements),
	SZ_VEC_BYTES(numElements * sizeof(float)),
	m_err(hipSuccess),
	h_A(SZ_VEC_BYTES), h_B(SZ_VEC_BYTES), h_C(SZ_VEC_BYTES, 0),
	d_A(nullptr), d_B(nullptr), d_C(nullptr)
{
	std::cout << "[Vector addition of " << numElements << " elements]\n";

	// Allocate the device input vectors
	m_err = hipMalloc((void **)&d_A, SZ_VEC_BYTES);
	if (m_err != hipSuccess)
	{
		throw std::runtime_error(_describe("Failed to allocate d_A"));
	}
	m_err = hipMalloc((void **)&d_B, SZ_VEC_BYTES);
	if (m_err != hipSuccess)
	{
		throw std::runtime_error(_describe("Failed to allocate d_B"));
	}
	m_err = hipMalloc((void **)&d_C, SZ_VEC_BYTES);
	if (m_err != hipSuccess)
	{
		throw std::runtime_error(_describe("Failed to allocate d_C"));
	}
}


VectorTester::~VectorTester()
{
	m_err = hipFree(d_A);
	if (m_err != hipSuccess)
	{
		throw std::runtime_error(_describe("Failed to free device vector A"));
	}

	m_err = hipFree(d_B);
	if (m_err != hipSuccess)
	{
		throw std::runtime_error(_describe("Failed to free device vector B"));
	}

	m_err = hipFree(d_C);
	if (m_err != hipSuccess)
	{
		throw std::runtime_error(_describe("Failed to free device vector C"));
	}
}


void VectorTester::randomize()
{
	for (int i = 0; i < m_numElements; i++)
	{
		h_A[i] = rand()/(float)RAND_MAX;
		h_B[i] = rand()/(float)RAND_MAX;
	}
}


void VectorTester::load_values()
{
	printf("Copy input data from the host memory to the CUDA device\n");

	m_err = hipMemcpy(d_A, h_A.data(), SZ_VEC_BYTES, hipMemcpyHostToDevice);
	if (m_err != hipSuccess)
	{
		throw std::runtime_error(_describe("Failed to copy vector A from host to device"));
	}

	m_err = hipMemcpy(d_B, h_B.data(), SZ_VEC_BYTES, hipMemcpyHostToDevice);
	if (m_err != hipSuccess)
	{
		throw std::runtime_error(_describe("Failed to copy vector B from host to device"));
	}
}


void VectorTester::check()
{
	printf("Copy output data from the CUDA device to the host memory\n");

	m_err = hipMemcpy(h_C.data(), d_C, SZ_VEC_BYTES, hipMemcpyDeviceToHost);

	if (m_err != hipSuccess)
	{
		throw std::runtime_error(_describe("Failed to copy vector C from device to host"));
	}

	// Verify that the result vector is correct
	for (int i = 0; i < m_numElements; ++i)
	{
		if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
		{
			throw std::runtime_error("Result verification failed at element %d " + std::to_string(i));
		}
	}

	printf("Test PASSED\n");
}


void VectorTester::launch()
{
	const int threadsPerBlock = 256;
	const int blocksPerGrid =(m_numElements + threadsPerBlock - 1) / threadsPerBlock;
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	d_vector_add<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, m_numElements);

	m_err = hipGetLastError();
	if (m_err != hipSuccess)
	{
		throw std::runtime_error(_describe("Failed to launch vectorAdd kernel"));
	}
}


void VectorTester::cpu_benchmark()
{
	for (int i = 0; i < m_numElements; i++)
	{
		h_C[i] = h_A[i] + h_B[i];
	}
}


std::string VectorTester::_describe(const std::string & prefix)
{
	return prefix + ": " + std::string(hipGetErrorString(m_err));
}



namespace pietra
{
	namespace cuda
	{
		/**
		 * Vector addition: C = A + B.
		 *
		 * This sample is a very basic sample that implements element by element
		 * vector addition. It is the same as the sample illustrating Chapter 2
		 * of the programming guide with some additions like error checking.
		 */
		int vector_test(int numElements)
		{
			pietra::utils::Performances performances;
			ADD_TIC

			VectorTester vTester(numElements);

			ADD_TIC

			vTester.randomize();

			ADD_TIC

			vTester.load_values();

			ADD_TIC

			vTester.launch();

			ADD_TIC

			vTester.cpu_benchmark();

			ADD_TIC

			vTester.check();

			ADD_TIC

			performances.tictoc_results();

			return 0;
		}
	}
}
