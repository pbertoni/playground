#include "hip/hip_runtime.h"
/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <thrust/device_ptr.h>
#include <thrust/count.h>
#include <thrust/execution_policy.h>

#include <iostream>
#include <chrono>
#include <vector>
#include <string>
#include <unistd.h>


#include <hip/hip_runtime_api.h>


/////////////////////////////////////////////////////////////////
// Some utility code to define grid_stride_range
// Normally this would be in a header but it's here
// for didactic purposes. Uses 
#include "range.hpp"

using namespace util::lang;



#include <sys/stat.h>

off_t fsize(const char * FILENAME)
{
	struct stat st;

	if (stat(FILENAME, &st) == 0)
	{
		return st.st_size;
	}
	else
	{
		return -1;
	}
}


/**
 * type alias to simplify typing...
 */
template<typename T>
using step_range = typename range_proxy<T>::step_range_proxy;


/**
 *
 */
template <typename T>
__device__
step_range<T> grid_stride_range(T begin, T end)
{
	begin += blockDim.x * blockIdx.x + threadIdx.x;
	return range(begin, end).step(gridDim.x * blockDim.x);
}


/**
 *
 */
template <typename T, typename Predicate>
__device__ 
void ours_count_if(int * count, T * data, int n, Predicate p)
{ 
	for (const auto i : grid_stride_range(0, n))
	{
		if (p(data[i]))
		{
			atomicAdd(count, 1);
		}
	}
}


/**
 * Use count_if with a lambda function that searches for x, y, z or w
 * Note the use of range-based for loop and initializer_list inside the functor
 * We use auto so we don't have to know the type of the functor or array
 */
__global__
void d_xyzw_frequency_ours(int * count, char * text, int n)
{
	const char letters[] { 'x','y','z','w'};

	ours_count_if(count, text, n,
			[&](char c)
			{
				for (const auto x : letters)
				{
					if (c == x)
					{
						return true;
					}
				}
				return false;
			});
}


/**
 *
 */
__global__
void d_xyzw_frequency_thrust(int *count, char *text, int n)
{
	const char letters[] = { 'x','y','z','w'};

	*count = thrust::count_if(thrust::device, text, text+n,
			[=](char c)
			{
				for (const auto x : letters)
				{
					if (c == x)
					{
						return true;
					}
				}
				return false;
			});
}


/**
 * a bug in Thrust 1.8 causes warnings when this is uncommented
 * so commented out by default -- fixed in Thrust master branch
 */
#define ALLOW_THRUST_HOST
#ifdef ALLOW_THRUST_HOST
void h_xyzw_frequency_thrust(int * count, char * text, int n)
{
	const char letters[] = {'x','y','z','w'};

	*count = thrust::count_if(thrust::host, text, text + n,
			[&](char c)
			{
				for (const auto x : letters)
				{
					if (c == x)
					{
						return true;
					}
				}
				return false;
			});
}
#endif


std::vector<std::pair<std::chrono::time_point<std::chrono::high_resolution_clock>, int>> vtp;


void add_tic(int lbl)
{
	vtp.emplace_back(std::chrono::high_resolution_clock::now(), lbl);
}

/*
void add_tic(int cazz)
{
	std::cerr << "cazz " << cazz;
}*/

#define ADD_TIC add_tic(__LINE__);


void tictoc_results()
{
	std::cout << "TIC-TOCS [us]";
	for (int t=0; t<vtp.size()-1; t++)
	{
		const auto TOC = vtp[t+1].first, TIC = vtp[t].first;
		const auto TOCLBL = vtp[t+1].second, TICLBL = vtp[t].second;
		std::cout << "\n\t" << t << " - lines " << TICLBL << " to " << TOCLBL
				<< " -> " << std::chrono::duration_cast<std::chrono::microseconds>(TOC-TIC).count();
	}
	std::cout << std::endl;
}


int main(int argc, char ** argv)
{ 
	const char * FILENAME = sdkFindFilePath(argv[1], argv[0]);

	// find first CUDA device
	int devID = findCudaDevice(argc, (const char **)argv);

	// Get file size
	const auto FILE_SZ_BYTES = fsize(FILENAME);

	// Allocate buffers
	char * h_text = new char[FILE_SZ_BYTES];
	char * d_text;
	checkCudaErrors(hipMalloc((void**)&d_text, FILE_SZ_BYTES));

	// Store file into buffer
	FILE  *fp = fopen(FILENAME, "r");
	if (fp == NULL)
	{
		printf("%s cannot find the input text file\n. Exiting..\n", argv[0]);
		return EXIT_FAILURE;
	}
	const int FILE_SZ_READ = fread(h_text, sizeof(char), FILE_SZ_BYTES, fp);
	fclose(fp);
	std::cout << "Read " << FILE_SZ_READ << " B from " << FILENAME << "(" << FILE_SZ_BYTES << " B)" << std::endl;

																ADD_TIC
	checkCudaErrors(hipMemcpy(d_text, h_text, FILE_SZ_READ, hipMemcpyHostToDevice));

	int count = 0;
	int * d_count;
																ADD_TIC
	checkCudaErrors(hipMalloc(&d_count, sizeof(int)));									// NEGLIGIBLE
	checkCudaErrors(hipMemset(d_count, 0, sizeof(int)));								// NEGLIGIBLE

	// GPU BURST
																ADD_TIC

	d_xyzw_frequency_ours<<<8, 256>>>(d_count, d_text, FILE_SZ_READ);					// NEGLIGIBLE

																ADD_TIC

	d_xyzw_frequency_thrust<<<1, 1>>>(d_count, d_text, FILE_SZ_READ);					// NEGLIGIBLE

																ADD_TIC

	checkCudaErrors(hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost));	// HEAVY

																ADD_TIC

	h_xyzw_frequency_thrust(&count, h_text, FILE_SZ_READ);								// E+3 magnitude
																ADD_TIC

	tictoc_results();

	std::cout << "Counted " << count << " instances of 'x', 'y', 'z', or 'w' in \"" << FILENAME << "\"" << std::endl;

	checkCudaErrors(hipFree(d_count));
	checkCudaErrors(hipFree(d_text));
	delete[] h_text;

	return EXIT_SUCCESS;
}
